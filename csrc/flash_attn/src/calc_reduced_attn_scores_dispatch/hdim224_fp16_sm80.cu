#include "launch_template.h"

namespace reduced_scores {
template<>
void run_<cutlass::half_t, 224>(Params &params, hipStream_t stream) {
    run_hdim224<cutlass::half_t>(params, stream);
}
} // namespace reduced_scores