#include "launch_template.h"

namespace reduced_scores {
template<>
void run_<cutlass::bfloat16_t, 32>(Params &params, hipStream_t stream) {
    run_hdim32<cutlass::bfloat16_t>(params, stream);
}
} // namespace reduced_scores