#include "launch_template.h"

namespace reduced_scores {
template<>
void run_<cutlass::bfloat16_t, 256>(Params &params, hipStream_t stream) {
    run_hdim256<cutlass::bfloat16_t>(params, stream);
}
} // namespace reduced_scores