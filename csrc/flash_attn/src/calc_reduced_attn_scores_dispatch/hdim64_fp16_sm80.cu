#include "launch_template.h"

namespace reduced_scores {
template<>
void run_<cutlass::half_t, 64>(Params &params, hipStream_t stream) {
    run_hdim64<cutlass::half_t>(params, stream);
}
} // namespace reduced_scores